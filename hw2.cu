#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  int col = threadIdx.x;
  // int row = threadIdx.y + blockIdx.y * blockDim.y;
  float runningSum = 0.0;

  for(int i = 0; i < numCRows; ++i) {
      for(int j = 0; j < numAColumns; ++j) {
        runningSum += (A[i * numAColumns + j] * B[j * numBColumns + col]);
      }
      C[i * numCColumns + col] = runningSum;   
      runningSum = 0.0;   
  }

  // if(col < numCColumns && row < numCRows) {
  //   for(int j = 0; j < numAColumns; ++j) {
  //     runningSum += A[row * numAColumns + j] * B[j * numBColumns + col];
  //   }
  //   C[row * numCColumns + col] = runningSum;
  // }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCColumns * numCRows * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc(&deviceA, numAColumns * numARows * sizeof(float));
  hipMalloc(&deviceB, numBColumns * numBRows * sizeof(float));
  hipMalloc(&deviceC, numCColumns * numCRows * sizeof(float));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numAColumns * numARows * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBColumns * numBRows * sizeof(float), hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  // int blocksPerGrid = 1024/numCColumns;
  int blocksPerGrid = 1;
  if(blocksPerGrid == 0) {
    blocksPerGrid++;
  }
  int threadsPerBlock = numCColumns;
  wbLog(TRACE, "BPG: ", blocksPerGrid);
  wbLog(TRACE, "TPB: ", threadsPerBlock);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCColumns * numCRows * sizeof(float), hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
